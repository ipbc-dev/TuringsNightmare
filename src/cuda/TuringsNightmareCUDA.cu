#include "hip/hip_runtime.h"
/*
Copyright 2018 Interplanetary Broadcast Coin SL

This file is part of Turings Nightmare
Authors: Fritjof Harms, Markus Behm

Permission is hereby granted, free of charge, to any person obtaining
a copy of this software and associated documentation files (the
"Software"), to deal in the Software without restriction, including
without limitation the rights to use, copy, modify, merge, publish,
distribute, sublicense, and/or sell copies of the Software, and to
permit persons to whom the Software is furnished to do so, subject to
the following conditions:

The above copyright notice and this permission notice shall be
included in all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND
NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE
LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION
OF CONTRACT, TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION
WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/
#include "cuda/TuringsNightmareCUDA.h"

#include <cstring>

// TODO: cleanup utility dependencies
#include <chrono>
#include <iostream>

#define ENTANGLED_UINT8 (uint8_t)(state->step_counter ^ state->register_a ^ state->register_b ^ state->register_c ^ state->register_d ^ state->hs.w[state->step_counter % 25] ^ state->hs.b[state->step_counter % 200] ^ state->step_limit ^ state->input_size)
#define ENTANGLED_UINT32 (uint32_t)(state->step_counter ^ state->register_a ^ state->register_b ^ state->register_c ^ state->register_d ^ state->hs.w[state->step_counter % 25] ^ state->hs.b[state->step_counter % 200] ^ state->step_limit ^ state->input_size)
#define ENTANGLED_UINT64 (uint64_t)(state->step_counter ^ state->register_a ^ state->register_b ^ state->register_c ^ state->register_d ^ state->hs.w[state->step_counter % 25] ^ state->hs.b[state->step_counter % 200] ^ state->step_limit ^ state->input_size)

__device__
inline uint8_t* TN_AtRelPos(VM_State *state, int position) {
	uint64_t pos = state->instruction_ptr + position;
	if (pos >= state->memory_size) pos %= state->memory_size;
	return state->memory + pos;
}

#define MEM(relpos) *TN_AtRelPos(state, relpos)

__device__
inline void TN_AdjustCycleLimit(VM_State *state, int change) {
	state->step_limit += change;

	if (state->step_limit < state->step_limit_min) state->step_limit = state->step_limit_min;
	else if (state->step_limit > state->step_limit_max) state->step_limit = state->step_limit_max;
}

#define MODCYCLES(change) TN_AdjustCycleLimit(state, change)

__device__
inline void TN_ParseInstruction(VM_State *state, VM_Instruction inst) {
	switch (inst) {
	case XOR:
		MEM(0) ^= MEM(MEM(MEM(-1)));
		break;
	case XOR2:
		MEM(1) ^= MEM(2);
		MEM(0) ^= MEM(1);
		break;
	case XOR3:
		MEM(0) ^= MEM(ENTANGLED_UINT8);
		break;
	case DIV:
		MEM(0) = MEM(0) ^ (MEM(1) / (MEM(ENTANGLED_UINT8) + 1));
		break;
	case ADD:
		MEM(1) += MEM(2);
		MEM(0) += MEM(1);
		break;
	case SUB:
		MEM(1) -= MEM(2);
		MEM(0) -= MEM(1);
		break;
	case INSTPTR:
		state->instruction_ptr = state->instruction_ptr * ENTANGLED_UINT64 % state->memory_size;
		break;
	case JUMP:
		state->instruction_ptr = ((state->instruction_ptr + ((ENTANGLED_UINT8 % 200) - 100)) % state->memory_size);
		break;
	case REGA_XOR:
		MEM(0) ^= state->register_a;
		state->register_a ^= MEM(MEM(1)) ^ ENTANGLED_UINT64;
		break;
	case REGB_XOR:
		MEM(0) ^= state->register_b;
		state->register_b ^= MEM(MEM(1)) ^ ENTANGLED_UINT64;
		break;
	case REGC_XOR:
		MEM(0) ^= state->register_c;
		state->register_c ^= MEM(MEM(1)) ^ ENTANGLED_UINT64;
		break;
	case REGD_XOR:
		MEM(0) ^= state->register_d;
		state->register_d ^= MEM(MEM(1)) ^ ENTANGLED_UINT64;
		break;
	case CYCLEADD:
		MODCYCLES(1 * ENTANGLED_UINT8);
		break;
	case CYCLESUB:
		MODCYCLES(-1 * ENTANGLED_UINT8);
		break;
	default:
	case NOOP:
		break;
	}
}

__device__
VM_Instruction TN_GetInstruction(VM_State *state) {
	return (VM_Instruction)((state->memory[state->instruction_ptr] ^ ENTANGLED_UINT64) % _LAST);
}

__global__
void TN_VM_Execute_CUDA(VM_State *mem) {
	VM_State *state = &mem[threadIdx.x];
	for (; state->step_counter <= state->step_limit; state->step_counter++) {
		VM_Instruction inst = TN_GetInstruction(state);
		TN_ParseInstruction(state, inst);
		state->instruction_ptr = (state->instruction_ptr + 1) % state->memory_size;
	}
}

void DeviceCUDA::run(const size_t N, VM_State *states) {
	VM_State *buf;
	hipMalloc((void**)&buf, sizeof(VM_State) * N);
	hipMemcpy(buf, states, sizeof(VM_State) * N, hipMemcpyHostToDevice);

	dim3 dimBlock(N, 1);
	dim3 dimGrid(1, 1);

	TN_VM_Execute_CUDA<<<dimGrid, dimBlock>>>(buf);

	hipDeviceSynchronize();
	hipMemcpy(states, buf, sizeof(VM_State) * N, hipMemcpyDeviceToHost);
	hipFree(buf);
}